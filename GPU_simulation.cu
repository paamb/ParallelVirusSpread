#include <bitset>
#include <iostream>
#include <random>
#include <cstring>
#include <chrono>
#include <utility>
#include <ostream>
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>
// For random numbers
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define NUM_COLS 64
#define INFECTIONRATE 0.1
#define GRID_DIM 64
#define BLOCK_DIM 32

int num_rows, num_healthy, num_infected;
float num_rows_normalizer;


// Init device variables
__device__ int d_num_rows, d_num_healthy, d_num_infected;
__device__ float d_num_rows_normalizer;
__device__ hiprandState cuda_random_value_state;

using namespace std;

/*

    Project in COSC3500. 
    Virus spread simulation.

    CUDA version
*/
uint64_t create_64_bit_infectionrate_bitmask(){
    uint64_t random_bit_string = 0;

    for (int i = 0; i < NUM_COLS*INFECTIONRATE; i++){
            int mask = 1ULL << i; 
            random_bit_string |= mask;
    }

    for(int i = 0; i < NUM_COLS*INFECTIONRATE; i++){

        uint64_t current_bit = 1LL << i;
    
        int random_index = rand() % (NUM_COLS);
        uint64_t random_bit = 1LL << random_index;


        if (!(random_bit & random_bit_string)){
            random_bit_string |= random_bit;
            uint64_t neg_current_bit = ~current_bit;
            random_bit_string &= neg_current_bit;  
        }


    }
    return random_bit_string;
}

void pretty_print_bit_matrix(uint64_t* grid){
    int total_ones = 0;
    for(int i = 0; i < num_rows; i++){
        bitset<NUM_COLS> bits(grid[i]);
        cout << bits;
        int ones = bits.count();
        cout << "    " << ones << endl;
        total_ones += ones;
    }

    cout << total_ones << endl;
}
/*

@brief Populates the grid. 


Example of what grid could look like after this step:  
00111
00111
00011
00011

The population have not been shuffled in this function

@param grid A pointer to the grid
@param num_bits_to_place The total number of 1's that should be placed on the grid.

*/
__global__ void populate_grid(uint64_t* grid, int num_bits_to_place){
    // Sets the see. Also include the index to not have the same seed every time
    int idx = blockIdx.x*BLOCK_DIM + threadIdx.x;
    int number_of_threads = GRID_DIM * BLOCK_DIM;
    //Initialize the grids to only contain 0's

    // Spread out equal amount of 1's on each row. The bits are not shuffled here.
    int extra_bits = num_bits_to_place % d_num_rows;
    for(int row = idx; row < d_num_rows; row+=number_of_threads){
        int bits_to_be_placed_in_row = num_bits_to_place * d_num_rows_normalizer;

        // For the first extra_bits rows. We want to add a bit.
        // This is different from the other versions as it is modified for CUDA
        if (row < extra_bits ){
            bits_to_be_placed_in_row++;
        } 
        // Set the bits from least significant bit
        uint64_t mask = (1 << bits_to_be_placed_in_row) - 1;
        grid[row] |= mask;
    }
}

/*

@brief Moves the population in the healthy grid. Each bit will either move to an empty spot, or stay where it is.

*/
__device__ void move_healthy(uint64_t* d_healthy_grid, int thread_index, int num_threads, int iteration){
    // Finds a healthy bit at a position. Moves it to another position if that is not empty.
    // Cuda random value generator
    hiprandState local_state;
    hiprand_init(thread_index * iteration + thread_index, iteration, 0, &local_state);

    for (int i = thread_index; i < d_num_rows; i+= num_threads){
        int number_of_ones = 0; 
        for (int j = 0; j < NUM_COLS; j++){
            uint64_t current_bit = 1LL << j;
            // If the current bit is 0. Then break. All trailing bits will be 0 aswell no need to move them. 
            if(!(d_healthy_grid[i] & current_bit)){break;}
            int random_index = __double2int_rn(hiprand_uniform(&local_state)*63.0f);
            // Shift a 1 by a random index. The position the '1'-bit should be moved to.
            uint64_t random_bit = 1LL << random_index;

            // If the position is empty then move the bit from pos = j to pos = random_index.

            if (!(random_bit & d_healthy_grid[i])){
                number_of_ones++;
                d_healthy_grid[i] |= random_bit;
                uint64_t neg_current_bit = ~current_bit;
                d_healthy_grid[i] = d_healthy_grid[i] & neg_current_bit;  
            }
        }
    }
}


/*

@brief Same function as move_healthy, but needs to take account for not moving to same place an healthy person aswell.

Made the extra function to aviod branching (if-statement to check if in infected or healthy grid)

*/
__device__ void move_infected(uint64_t* d_healthy_grid, uint64_t* d_infected_grid, int thread_index, int num_threads, int iteration){

    // Cuda random value generator
    hiprandState local_state;
    hiprand_init(thread_index * iteration + thread_index, iteration, 0, &local_state);
    // This does essentially the same as move_healthy, but it needs to check if of healthy individuals aswell.
    for (int i = thread_index; i < d_num_rows; i+=num_threads){
        for (int j = 0; j < NUM_COLS; j++){
            uint64_t current_bit = 1LL << j;
            if(!(d_infected_grid[i] & current_bit)) {break;};

            int random_index = __double2int_rn(hiprand_uniform(&local_state)*63.0f);
            // Find random index can only be moved inside the coloum (spatial locality).

            // Shift a 1 by a random index. The position the '1'-bit should be moved to.
            uint64_t random_bit = 1LL << random_index;

            // Make sure to not move to a position where a healthy person is
            if (!(random_bit & d_healthy_grid[i] || d_infected_grid[i] & random_bit)){
                d_infected_grid[i] |= random_bit;
                uint64_t neg_current_bit = ~current_bit;
                d_infected_grid[i] = d_infected_grid[i] & neg_current_bit;
            }
        }
    }
}


/**

Infect function. Healthy people will be infected by sick people around

**/
pair<int, int> infect(uint64_t* infected_grid, uint64_t* healthy_grid){
    


    uint64_t* healthy_grid_copy = (uint64_t*) malloc(num_rows* sizeof(uint64_t));
    uint64_t* infected_grid_copy = (uint64_t*) malloc(num_rows* sizeof(uint64_t));

    memcpy(healthy_grid_copy, healthy_grid, num_rows*sizeof(uint64_t));
    memcpy(infected_grid_copy, infected_grid, num_rows*sizeof(uint64_t));



    int num_infected = 0;
    int num_healthy = 0;

    // Random bit string with 
    uint64_t possible_infect_positions = create_64_bit_infectionrate_bitmask();

    for(int i = 0; i < num_rows; i++){
        // Infect vertically
        // Loop through directions up and down
        uint64_t curr_infection_row = infected_grid[i];
        for(int j = i - 1; j <= i + 1; j++){
            
            // Skipping the top, bottom and the same row for vertically infection
            if (j < 0 || j == i || j >= num_rows){ continue;}

            // Find the people that are up or down for a infected_person
            uint64_t infect_vertically = curr_infection_row & healthy_grid[j];

            // Infect with a rate of INFECTIONRATE
            infect_vertically = infect_vertically & possible_infect_positions;
            
            //Remove healthy people from the grid. By doing NAND-ing with the infected uint64
            healthy_grid_copy[j] &= ~infect_vertically;
            
            // Add the infected people to the infected_grid by OR-ing
            infected_grid_copy[j] |= infect_vertically;
        }

        // Infect to the left and right
        uint64_t left_shift = infected_grid[i] << 1;
        uint64_t right_shift = infected_grid[i] >> 1;
        

        // Shift both ways. Equivalent to infecting both right and left.
        // These are the people that can possibly be infected.
        uint64_t possible_infections_horizontally = (left_shift | right_shift) & healthy_grid[i];
        
        uint64_t actual_infections_horizontally = 0;
        actual_infections_horizontally = possible_infections_horizontally & possible_infect_positions; 


        // Remove from healthy
        healthy_grid_copy[i] &= ~actual_infections_horizontally;

        // Add to infected
        infected_grid_copy[i] |= actual_infections_horizontally;

        // Count number of infected people in the row
        bitset<NUM_COLS> infected_bits(infected_grid_copy[i]);
        num_infected += infected_bits.count();

        // Count number of infected people in the row
        bitset<NUM_COLS> healthy_bits(healthy_grid_copy[i]);
        num_healthy += healthy_bits.count();
    }

    // Copy over the two grids.
    memcpy(healthy_grid, healthy_grid_copy, num_rows*sizeof(uint64_t));
    memcpy(infected_grid, infected_grid_copy, num_rows*sizeof(uint64_t));

    free(healthy_grid_copy); 
    free(infected_grid_copy);

    return make_pair(num_infected, num_healthy);

}

// Set global HOST parameters
void set_global_parameters(){
    num_healthy = num_rows*NUM_COLS*0.2;
    num_infected = num_rows;
    num_rows_normalizer = 1.0/num_rows;
}

// Set global DEVICE parameters
void set_cuda_parameters(){
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_rows), &num_rows, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_rows_normalizer), &num_rows_normalizer, sizeof(int), 0, hipMemcpyHostToDevice);
}

__global__ void move(uint64_t *d_healthy_grid, uint64_t *d_infected_grid, int iteration){

    int idx = blockIdx.x*BLOCK_DIM + threadIdx.x;
    int number_of_threads = GRID_DIM * BLOCK_DIM;
    move_healthy(d_healthy_grid, idx, number_of_threads, iteration);
    move_infected(d_healthy_grid, d_infected_grid, idx, number_of_threads, iteration);
}


int main(int argc, char *argv[]){

    if (argc > 1) {
        num_rows = atoi(argv[1]);
    } else {
        num_rows = 8;
    };


    cout << num_rows << endl; 

    set_global_parameters();
    set_cuda_parameters();
    hipDeviceSynchronize();

    int healthy_people = num_healthy;
    int infected_people = num_infected;

    // Define the two grids. Each spot is represented as bits.
    // For the healthy_grid: HEALTHY = 1, EMPTY = 0
    uint64_t* healthy_grid = new uint64_t[num_rows];
    // For the infected_grid: INFECTED = 1, EMPTY = 0
    uint64_t* infected_grid = new uint64_t[num_rows];

    // Define the device grids
    uint64_t *d_healthy_grid, *d_infected_grid;

    // Allocate memory for the device healthy and infected matrices
    hipMalloc((void**) &d_healthy_grid, num_rows*sizeof(uint64_t));
    hipMalloc((void**) &d_infected_grid, num_rows*sizeof(uint64_t));

    // Define copy_device grids. Used in infect function.
    uint64_t *d_healthy_grid_copy, *d_infected_grid_copy;
    hipMalloc((void**) &d_healthy_grid_copy, num_rows*sizeof(uint64_t));
    hipMalloc((void**) &d_infected_grid_copy, num_rows*sizeof(uint64_t));


    cout << "Infected at start: " << infected_people << endl;
    cout << "Healthy  at start: " << healthy_people << endl;
    cout << endl;
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_healthy), &healthy_people, sizeof(healthy_people), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_num_infected), &infected_people, sizeof(infected_people), 0, hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < 100; i++){
        hipMemset(d_healthy_grid, 0, num_rows * sizeof(uint64_t));
        hipMemset(d_infected_grid, 0, num_rows * sizeof(uint64_t));
        
        // Populate the grids on GPU
        populate_grid<<<GRID_DIM,BLOCK_DIM>>>(d_healthy_grid, healthy_people);
        populate_grid<<<GRID_DIM,BLOCK_DIM>>>(d_infected_grid, infected_people);

        // Move both infected and healthy people on GPU
        move<<<GRID_DIM, BLOCK_DIM>>>(d_healthy_grid, d_infected_grid, i);
        hipMemcpy(healthy_grid, d_healthy_grid, num_rows*sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(infected_grid, d_infected_grid, num_rows*sizeof(uint64_t), hipMemcpyDeviceToHost);

        // Infect and find number of infected and healthy people on CPU
        pair<int, int> res = infect(infected_grid, healthy_grid);
        infected_people = res.first;
        healthy_people = res.second;
        hipMemcpyToSymbol(HIP_SYMBOL(d_num_healthy), &healthy_people, sizeof(healthy_people), 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL(d_num_infected), &infected_people, sizeof(infected_people), 0, hipMemcpyHostToDevice);
    }
    

    // Copy healthy people from GPU to print
    hipMemcpyFromSymbol(&healthy_people, HIP_SYMBOL(d_num_healthy), sizeof(healthy_people), 0, hipMemcpyDeviceToHost);
    hipMemcpyFromSymbol(&infected_people, HIP_SYMBOL(d_num_infected), sizeof(infected_people), 0, hipMemcpyDeviceToHost);
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end-start);
    cout << "Duration: " << duration.count() << endl;

    cout << "Infected at end: " << infected_people << endl;
    cout << "Healthy  at end: " << healthy_people << endl;
        int sum_inf_health = infected_people + healthy_people;
    cout << "Infected + healthy " <<sum_inf_health << endl;

    
    delete[] healthy_grid;
    delete[] infected_grid;
    hipFree(d_healthy_grid);
    hipFree(d_infected_grid);
    return 0;
}